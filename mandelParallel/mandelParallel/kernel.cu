﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <chrono>

// Testing 

#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

constexpr int X = 1920;
constexpr int Y = 1080;

__device__ int xParam = X;
__device__ int yParam = Y;

struct Point {
    int x;
    int y;
    int iteration;
    int sizeX;
    int sizeY;
};

hipError_t mandelBrotCalc(struct Point* pointArray, int* numIterations, unsigned long size);

__global__ void computeSet(struct Point* returnPointArr, int* numIterations) {   

    // Calculates Strides by finding the area a block is supposed to work on and add an offset based on the thread ID.
    //                              BEGINNING OF BLOCK                 THREAD OFFSET
    int block_xStart    = xParam * ((double) blockIdx.x / gridDim.x) + xParam * threadIdx.x / blockDim.x / gridDim.x;
    int block_xEnd      = xParam * ((double) blockIdx.x / gridDim.x) + xParam * (threadIdx.x + 1) / blockDim.x / gridDim.x;

    int block_yStart    = yParam * ((double) blockIdx.y / gridDim.y) + yParam * threadIdx.y / blockDim.y / gridDim.y;
    int block_yEnd      = yParam * ((double) blockIdx.y / gridDim.y) + yParam * (threadIdx.y + 1) / blockDim.y / gridDim.y;
    
    if (blockIdx.x == 5 && blockIdx.y == 5) {
            printf("Block %d, %d\tThread: %d, %d\tStride: %d->%d; %d->%d\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, block_xStart, block_xEnd, block_yStart, block_yEnd);
    }
    /*
    for (int i = block_xStart; i < block_xEnd; i++)
    {
        for (int j = block_yStart; j < block_yEnd; j++)
        {

            float x0 = i / (xParam * 2.47) - 2;
            float y0 = j / (yParam * 2.24) - 1.12;
            float x = 0.0;
            float y = 0.0;

            int iteration = 0;
            int max_iteration = 1000;

            while (x * x + y * y <= (2 * 2) && iteration < max_iteration)
            {
                float xtemp = x * x - y * y + x0;
                y = 2 * x * y + y0;
                x = xtemp;

                iteration++;
            }

            struct Point newPoint = { i, j, iteration, 0, 0 };

            returnPointArr[i * yParam + j] = newPoint;

        }

    }*/
}

/*
*
* Unoptimized escape for calculating the Mandelbrot Set
* 
*/
int main()
{
    // For Checking Devices in System ( Debugging mainly )
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device Number: " << i << std::endl;
        std::cout << "  Device name: " << prop.name << std::endl;
    }

    std::cout.setf(std::ios_base::unitbuf);

    struct Point *pointArray = new struct Point[X * Y];
    
    int* numIterations = new int[X * Y];

    std::cout << "Beginning Calculation" << std::endl;

    // Add vectors in parallel.
    hipError_t cudaStatus = mandelBrotCalc(pointArray, numIterations, (unsigned long) X * Y );
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mandelBrotCalc failed!");
        return 1;
    }

    std::cout << "Ending Calculation successfully, Beginning file output" << std::endl;
    
    // For Desktop
    std::ofstream file("C:\\Users\\tasma\\Desktop\\Textbooks\\mandelCuda\\CSVOutputs\\MandelSetOut_Parallel.csv");

    // For Laptop
    // std::ofstream file("C:\\Users\\Devil\\Desktop\\Random Docs\\CudaProgramming\\mandelCuda\\CSVOutputs\\mandelParallel.csv");

    file << "X,Y,Iteration,sizeX,sizeY" << std::endl;

    for (int i = 0; i < X * Y; i++) {
        if (i == 0) {
            file << pointArray[i].x << "," << pointArray[i].y << "," << pointArray[i].iteration << ","
                << X << ","
                << Y << "," << std::endl;
        }

        file << "," << pointArray[i].x << "," << pointArray[i].y << "," <<   pointArray[i].iteration << std::endl;
    }

    // Output results
    std::cout << "Ending file output" << std::endl;

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to calculate mandelBrot set in parallel.
hipError_t mandelBrotCalc (struct Point* pointArray, int* numIterations, unsigned long size)
{
    struct Point *dev_points = 0;
    int* dev_iterations = 0;

    dim3 nthreads(16, 9);
    dim3 nblocks(6, 6);

    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for output vectors (two input, one output).
    cudaStatus = hipMalloc((void**)&dev_points, X * Y * sizeof(struct Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_iterations, X * Y * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Launch a kernel on the GPU with 16 threads for each element.
    // Num blocks, numThreads
    /*
        Notes:
        blockDim.x,y,z gives the number of threads in a block, in the particular direction
        gridDim.x,y,z gives the number of blocks in a grid, in the particular direction
        blockDim.x * gridDim.x gives the number of threads in a grid (in the x direction, in this case)

    */
    
    // Use Grid Dim to define grid paramters
    // In kernel, use the block Id and the x, y thread indices to find which section of the 
    // Array the thread should run over

    computeSet CUDA_KERNEL (nblocks, nthreads)  (dev_points, dev_iterations);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pointArray, dev_points, X * Y * sizeof(struct Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy points failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(numIterations, dev_iterations, X * Y * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy iterations failed!");
        goto Error;
    }

Error:
    hipFree(dev_points);
    
    return cudaStatus;
}
