#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <chrono>


#ifdef __INTELLISENSE__
#define CUDA_KERNEL(...)
#else
#define CUDA_KERNEL(...) <<< __VA_ARGS__ >>>
#endif

#define X 1920
#define Y 1080

#define numBlocks 16 
#define numThreads 16 

struct Point {
    int x;
    int y;
    int iteration;
    int sizeX;
    int sizeY;
};

hipError_t mandelBrotCalc(struct Point* pointArray, int* numIterations, unsigned long size);

__global__ void computeSet(struct Point* returnPointArr, int* numIterations) {
    int iterations = 0;

    for (int i = X / threadIdx.x; i < X && i < X / (threadIdx.x + 1); i++) {
        for (int j = Y / threadIdx.y; j < Y && j < Y / (threadIdx.y + 1); j++) {

            float x0 = i / (double)X * 2.47 - 2;
            float y0 = j / (double)Y * 2.24 - 1.12;
            float x = 0.0;
            float y = 0.0;

            int iteration = 0;
            int max_iteration = 1000;

            while (x * x + y * y <= (2 * 2) && iteration < max_iteration) {
                float xtemp = x * x - y * y + x0;
                y = 2 * x * y + y0;
                x = xtemp;

                iteration++;
            }

            struct Point newPoint = { i, j, iteration, 0, 0 };

            returnPointArr[];

            iterations++;
        }

        iterations++;
    }

    numIterations[threadIdx.x / numThreads + threadIdx.y % 16] = iterations;
}

/*
*
* Unoptimized escape for calculating the Mandelbrot Set
* 
*/
int main()
{
    struct Point pointArray[X * Y];
    
    int* numIterations = (int*) malloc(sizeof(int) * numThreads * numBlocks);

    // Add vectors in parallel.
    hipError_t cudaStatus = mandelBrotCalc(pointArray, numIterations, (unsigned long) X * Y );
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    std::ofstream file("MandelSetOut_Parallel.csv");

    file << "X,Y,Iteration,sizeX,sizeY" << std::endl;

    for (int i = 0; i < X * Y; i++) {
        if (i == 0) {
            file << pointArray[i].x << "," << pointArray[i].y << "," << pointArray[i].iteration
                << X << ","
                << Y << "," << std::endl;
        }

        file << pointArray[i].x << "," << pointArray[i].y << "," << pointArray[i].iteration << std::endl;
    }

    // Output results

    

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to calculate mandelBrot set in parallel.
hipError_t mandelBrotCalc (struct Point pointArray[], int* numIterations, unsigned long size)
{
    struct Point *dev_points = 0;
    int* dev_iterations = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for output vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_points, size * sizeof(struct Point));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_iterations, numThreads * numBlocks * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Launch a kernel on the GPU with 16 threads for each element.
    // Num blocks, numThreads
    CUDA_KERNEL (16, 16) computeSet (dev_points, dev_iterations);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(&pointArray, dev_points, size * sizeof(struct Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(&numIterations, dev_iterations, size * sizeof(struct Point), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_points);
    
    return cudaStatus;
}
